#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h> 
#include <stdio.h>
#include <time.h>
#include <math.h>

void MatMulNoTiling(float* A, int Ax, int Ay, float* B, int Bx, int By);
void MatMulWithTiling(float* A, int Ax, int Ay, float* B, int Bx, int By);
void printValues(float* A, int Ax, int Ay);
__global__ void MatMulNoTilingKer(float* d_A, float* d_B, float* d_res, int Bx, int Ay, int Ax);
__global__ void MatMulWithTilingKer(float* d_A, float* d_B, float* d_res, int Bx, int Ay, int Ax);

int main()
{

	//Initialize dimensions of input arrays
	int Ax = 3;
	int Ay = 2;

	int Bx = 4;
	int By = 3;

	if (Ax != By)
		printf("Ax must be equal to By");

	float* A = (float*)malloc((Ax * Ay) * sizeof(float));

	float* B = (float*)malloc((Bx * By) * sizeof(float));

	//Initialize values of A sequentially
	if (A != NULL) {
		for (int i = 0; i < (Ax * Ay); i++)
		{
			//Enter the numbers into A and B's elements
			A[i] = i + 1;
			
		}

	}

	//Initialize values of B sequentially
	if (B != NULL) {
		for (int i = 0; i < (Bx * By); i++)
		{
			//Enter the numbers into A and B's elements
			B[i] = i + 1;

		}

	}

	printValues(A, Ax, Ay);

	printf("\n Multiplied By \n\n");

	printValues(B, Bx, By);

	printf("\n");

	time_t start = clock();

	/*

	MatMulNoTiling(A, Ax, Ay, B, Bx, By);

	time_t passed = 1000 * (clock() - start) / CLOCKS_PER_SEC;

	printf("\nExecution time (ms) without tiling: %d", passed);

	*/

	start = clock();

	MatMulWithTiling(A, Ax, Ay, B, Bx, By);

	time_t passed = 1000 * (clock() - start) / CLOCKS_PER_SEC;

	printf("\nExecution time (ms) with tiling: %d", passed);

	free(A);
	free(B);

	return 0;

}


void printValues(float* A, int Ax, int Ay) {

	for (int y = 0; y < Ay; y++)
	{
		for (int x = 0; x < Ax; x++)
			printf("%9.6f \t", A[x + y * Ax]);


		printf("\n");

	}

}




void MatMulNoTiling(float* A, int Ax, int Ay, float* B, int Bx, int By) 
{

	//Declare result matrix
	float* res = (float*)malloc((Ay * Bx) * sizeof(float));

	//Allocate memory on device
	float* d_res, *d_A, *d_B;
	hipMalloc(&d_res, Ay * Bx * sizeof(float));
	hipMalloc(&d_A, Ax * Ay * sizeof(float));
	hipMalloc(&d_B, Bx * By * sizeof(float));

	//Copy values to device memory
	hipMemcpy(d_A, A, Ax * Ay * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, Bx * By * sizeof(float), hipMemcpyHostToDevice);



	//Get number of blocks in x direction
	int n_blocksx = ceil((double)Bx / 1024);

	int n_blocksy = Ay;

	const dim3 blockSize(1024, 1, 1);

	const dim3 gridSize(n_blocksx, n_blocksy, 1);


	MatMulNoTilingKer << <gridSize, blockSize >> > (d_A, d_B, d_res, Bx, Ay, Ax);
	

	hipMemcpy(res, d_res, Ay * Bx * sizeof(float), hipMemcpyDeviceToHost);

	printValues(res, Bx, Ay);

	free(res);

}


//Kernel function for matrix multiplication without tiling
__global__ void MatMulNoTilingKer(float* d_A, float* d_B, float* d_res, int Bx, int Ay, int Ax) {

	//Ax = By

	//Index in res
	int x = threadIdx.x + blockIdx.x * blockDim.x;

	int y = blockIdx.y;

	if (x < Bx && y < Ay)
	{

	float sum = 0;

	//Each thread calculate summation
	for (int i = 0; i < Ax; i++)
	{

		printf("(%d, %d) * (%d, %d) = %9.6f * %9.6f \n", y + 1, i + 1, i + 1, x + 1, d_A[i + y * Ax], d_B[x + Bx * i]);

		sum += d_A[i + y * Ax] * d_B[x + Bx * i];
	}

	d_res[x + y * Bx] = sum;

	}

}



void MatMulWithTiling(float* A, int Ax, int Ay, float* B, int Bx, int By) {



	//Declare result matrix
	float* res = (float*)malloc((Ay * Bx) * sizeof(float));

	//Allocate memory on device
	float* d_res, * d_A, * d_B;
	hipMalloc(&d_res, Ay * Bx * sizeof(float));
	hipMalloc(&d_A, Ax * Ay * sizeof(float));
	hipMalloc(&d_B, Bx * By * sizeof(float));

	//Copy values to device memory
	hipMemcpy(d_A, A, Ax * Ay * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, Bx * By * sizeof(float), hipMemcpyHostToDevice);



	//Get number of blocks in x direction
	int n_blocksx = ceil((double)Bx / 32);

	int n_blocksy = Ay/32;

	const dim3 blockSize(32, 32, 1);

	const dim3 gridSize(n_blocksx, n_blocksy, 1);


	MatMulWithTilingKer << <gridSize, blockSize >> > (d_A, d_B, d_res, Bx, Ay, Ax);

	hipMemcpy(res, d_res, Ay * Bx * sizeof(float), hipMemcpyDeviceToHost);

	printValues(res, Bx, Ay);

	free(res);

}






//Kernel function for matrix multiplication with tiling
__global__ void MatMulWithTilingKer(float* d_A, float* d_B, float* d_res, int Bx, int Ay, int Ax) {

	//Ax = By

	//Shared memory fits 2000 floats
	__shared__ float TileA[32][32];
	__shared__ float TileB[32][32];
	
	//Index in res
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	//Index for shared memory
	int my_x = threadIdx.x;
	int my_y = threadIdx.y;


	//Initialize tiles for A and B matrices
	if (x < Ax && y < Ay)
		TileA[my_x][my_y] = d_A[x + y * Ax];


	if (x < Bx && y < Ax)
		TileB[my_x][my_y] = d_B[y + x * Bx];

	__syncthreads();

	if (x < Bx && y < Ay)
	{

		float sum = 0;

		//Each thread calculate summation
		for (int i = 0; i < blockDim.x; i++)
		{

			printf("(%d, %d) * (%d, %d) = %9.6f * %9.6f \n", my_y + 1, i + 1, i + 1, my_x + 1, TileA[i][my_y], TileB[my_x][i]);

			sum += TileA[i][my_y] * TileB[my_x][i];

			//printf("(%d, %d) * (%d, %d) = %9.6f * %9.6f \n", y + 1, i + 1, i + 1, x + 1, d_A[i + y * Ax], d_B[x + Bx * i]);

			//sum += d_A[i + y * Ax] * d_B[x + Bx * i];


		}


		d_res[x + y * Bx] = sum;

	}



}